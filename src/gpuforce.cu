
#include <hip/hip_runtime.h>
#ifdef GPU

extern "C" {

#include <math.h>
#include <cuda.h>
#include <cuda_runtime_api.h>
#include <math.h>
#include "common.h"

}

#include "helper_cuda.h"


#define BLOCK_SIZE 256
#define SOFTENING 0.0f

double3 *pos_dev;
double3 *acc_dev;
double *masses_dev;

int inited = 0;



void calculate_force_cuda(double4* oldPos, double G, int numBodies, double4* acc);
__global__ void cudaforce(double4* oldPos, double G, int numBodies, double3* acc);
void integrateNbodySystem(double4 *dPos, double3 *acc,
                           double G,
                           unsigned int numBodies,
                           int blockSize);

template<typename T>
__global__ void gpuforce(double4 *p, T G, int n, double3 *acc) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        T Fx = 0.0f; T Fy = 0.0f; T Fz = 0.0f;

#pragma unroll
        for (int j = 0; j < n; j++) {
            T m = p[j].w;
            if (i == j || m == 0) continue;
            T dx = p[i].x - p[j].x;
            T dy = p[i].y - p[j].y;
            T dz = p[i].z - p[j].z;
            T distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            if (distSqr == SOFTENING) continue;
            T invDist = rsqrt(distSqr);
            T invDist3 = invDist * invDist * invDist;

            Fx -= (G * m * dx * invDist3);
            Fy -= (G * m * dy * invDist3);
            Fz -= (G * m * dz * invDist3);
        }
        acc[i].x = Fx; acc[i].y = Fy; acc[i].z = Fz;
    }
}

template<typename T>
__global__ void gpuforce_v2(double3 *p, T *masses, T G, int n, double3 *acc) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        T Fx = 0.0f; T Fy = 0.0f; T Fz = 0.0f;

#pragma unroll
        for (int j = 0; j < n; j++) {
            T m = masses[j];
            if (i == j || m == 0) continue;
            T dx = p[i].x - p[j].x;
            T dy = p[i].y - p[j].y;
            T dz = p[i].z - p[j].z;
            T distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            if (distSqr == SOFTENING) continue;
            T invDist = rsqrt(distSqr);
            T invDist3 = invDist * invDist * invDist;

            Fx -= (G * m * dx * invDist3);
            Fy -= (G * m * dy * invDist3);
            Fz -= (G * m * dz * invDist3);
        }
        acc[i].x = Fx; acc[i].y = Fy; acc[i].z = Fz;
    }
}

extern "C" {

    void gpu_init(int N, int deviceID) {
        printf("Device id, %d\n", deviceID);
        if (inited) return;
        int device_count = 0;
        cudaGetDeviceCount(&device_count);
        if (device_count == 0) {
            printf("No CUDA device found. Disable GPU acceleration...\n");
            sim.devID = -1;
        } else if (deviceID < device_count) {
            if (deviceID >= 0) {
                // initialize the GPU device 
                sim.devID = gpuDeviceInit(deviceID);
                printf("Device ID = %d, total number of GPU devices: %d\n", sim.devID, device_count);

                // allocate GPU memory
                int bytes = N * sizeof(double3);
                checkCudaErrors(cudaMalloc(&pos_dev, bytes));
                checkCudaErrors(cudaMalloc(&acc_dev, N * sizeof(double3)));
                checkCudaErrors(cudaMalloc(&masses_dev, N * sizeof(double)));

                inited = 1;
                printf("GPU force opened.\n");
            } else {
                // the user chooses to use CPU only because deviceID < 0
                printf("GPU acceleration disabled by the user (deviceID = %d)", deviceID);
                sim.devID = deviceID;
            }
        } else {
            printf("Invalid CUDA device ID. Number of devices: %d, given device ID: %d. Disable GPU acceleration...\n", device_count, deviceID);
            sim.devID = -1;
        }
    }

    void gpu_finalize() {
        printf("Closing CPU force...");
        if (pos_dev != NULL) cudaFree(pos_dev);
        if (acc_dev != NULL) cudaFree(acc_dev);
        printf("done.\n");
    }


    size_t ode_n_body_second_order_gpu(const real vec[], size_t N, real G, const real masses[], const real radii[], real acc[]) {
        if (masses == NULL) {printf("masses=NULL, exiting...\n"); exit(0);}
        double * pos_host = (double *)malloc(N * 3 * sizeof(double));
        //for (size_t i = 0; i < N; i++) {
        //    pos_host[3 * i] = vec[3 * i];
        //    pos_host[3 * i + 1] = vec[3 * i + 1];
        //    pos_host[3 * i + 2] = vec[3 * i + 2];
        //    // pos_host[4 * i + 3] = masses[i];
        //}

        cudaError_t err;


        checkCudaErrors(cudaMemcpy(pos_dev, vec, N*sizeof(double3), cudaMemcpyHostToDevice));
        checkCudaErrors(cudaMemcpy(masses_dev, masses, N*sizeof(double), cudaMemcpyHostToDevice));

        int actual_block_size = BLOCK_SIZE;
        int nBlocks = (N + actual_block_size - 1) / actual_block_size;

        gpuforce_v2<double><<<nBlocks, actual_block_size>>>(pos_dev, masses_dev, (double) G, (int) N, acc_dev);
        // integrateNbodySystem(pos_dev, acc_dev, G, N, BLOCK_SIZE);
        //cudaforce<<<nBlocks, actual_block_size, shm_size>>>(pos_dev, (double) G, (int) N, acc_dev);

        err = cudaGetLastError();
        if (err != cudaSuccess) {printf("Error: %d %s\n", err, cudaGetErrorString(err)); exit(0);}


        checkCudaErrors(cudaMemcpy(acc, acc_dev, N*sizeof(double3), cudaMemcpyDeviceToHost));
        /*
        for (size_t i = 0; i < 3*N; i++) {
            acc[i] = (real) acc_host[i];
        }*/
        

        // printf("\n");
        // for (int i = 0; i < 3 * N; i++) printf("%f\t", acc[i]);
        // exit(0);
        // for (int i = 0; i < 3 * N; i++) acc[i] = (real) acc_host[i];
        // free(pos_host);

        return 0;
    }
} // end extern C

#endif
